/* 
 * File:   MseErrorComputer.cpp
 * Author: janvojt
 * 
 * Created on November 29, 2014, 12:58 PM
 */

#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>


__global__
void sumVectors(double *dA, double *dB, int elements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < elements) {
        dA[i] += dB[i];
    }
}
void k_sumVectors(double *dA, double *dB, int elements) {
    int ts = 512;
    int bs = (elements + ts - 1) / ts;
    sumVectors<<<bs,ts>>>(dA, dB, elements);
}


__global__
void computeOutputLocalGradient(double *actualOutput, double *expectedOutput, double *localGradient, int elements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < elements) {
        double derivative = actualOutput[i] * (1.0 - actualOutput[i]);
        localGradient[i] = (actualOutput[i] - expectedOutput[i]) * derivative;
    }
}
void k_computeOutputLocalGradient(double *actualOutput, double *expectedOutput, double *localGradient, int elements) {
    int ts = 512;
    int bs = (elements + ts - 1) / ts;
    computeOutputLocalGradient<<<bs,ts>>>(actualOutput, expectedOutput, localGradient, elements);
}


__global__
void computeTotalDerivative(double learningRate, int nextNeurons,
        double *thisInput, double *nextLocalGradient,
        double *weightDiffs, int elements) {
    
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < elements) {
        int i = idx / nextNeurons;
        int j = idx % nextNeurons;
        weightDiffs[i*nextNeurons+j] = -learningRate * nextLocalGradient[j] * thisInput[i];
    }
}
void k_computeTotalDerivative(int thisNeurons, int nextNeurons, 
        double learningRate, double *thisInput, double *nextLocalGradient,
        double *weightDiffs) {
    int ts = 512;
    int bs = (thisNeurons * nextNeurons + ts - 1) / ts;
    computeTotalDerivative<<<bs,ts>>>(learningRate, nextNeurons,
        thisInput, nextLocalGradient,
        weightDiffs, thisNeurons * nextNeurons);
}


__global__
void computeBiasDerivative(double learningRate, double *nextLocalGradient,
        double *biasDiffs, int elements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < elements) {
        biasDiffs[i] = -learningRate * nextLocalGradient[i];
    }
}
void k_computeBiasDerivative(
        double learningRate, double *nextLocalGradient,
        double *biasDiffs, int elements) {
    int ts = 512;
    int bs = (elements + ts - 1) / ts;
    computeBiasDerivative<<<bs,ts>>>(learningRate, nextLocalGradient,
        biasDiffs, elements);
}


__global__
void computeHiddenLocalGradient(
        int thisNeurons, int nextNeurons,
        double *thisInput, double *weights,
        double *thisLocalGradient, double *nextLocalGradient) {
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i < thisNeurons) {
        double derivative = thisInput[i] * (1.0 - thisInput[i]);

        double sumNextGradient = 0;
        for (int j = 0; j<nextNeurons; j++) {
            sumNextGradient += nextLocalGradient[j] * weights[i * nextNeurons + j];
        }
        thisLocalGradient[i] = sumNextGradient * derivative;
    }
}
void k_computeHiddenLocalGradient(
        int thisNeurons, int nextNeurons,
        double *thisInput, double *weights,
        double *thisLocalGradient, double *nextLocalGradient) {
    
    int ts = 512;
    int bs = (thisNeurons + ts - 1) / ts;
    computeHiddenLocalGradient<<<bs,ts>>>(
        thisNeurons, nextNeurons,
        thisInput, weights,
        thisLocalGradient, nextLocalGradient);
}


__global__
void computeSigmoid(double *dArray, int elements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < elements) {
        dArray[i] = 1.0 / (1.0 + exp(-dArray[i]));
    }
}
void k_computeSigmoid(double *dArray, int elements) {
    int ts = 512;
    int bs = (elements + ts - 1) / ts;
	computeSigmoid<<<bs,ts>>>(dArray, elements);
}

__global__
void spreadInterval(double min, double max, double *dArray, int elements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < elements) {
        dArray[i] = (dArray[i] * (max - min)) + min;
    }
}
void k_spreadInterval(double min, double max, double *dArray, int size) {
    int ts = 512;
    int bs = (size + ts - 1) / ts;
    spreadInterval<<<bs,ts>>>(min, max, dArray, size);
}